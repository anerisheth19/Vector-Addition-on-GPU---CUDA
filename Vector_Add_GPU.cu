//ECGR 6090 Heterogeneous Computing Homework 0
// Problem 1 a - Vector Add on GPU
//Written by Aneri Sheth - 801085402

// Reference taken from Lecture Slides by Dr. Tabkhi 
// Other references taken from - http://ecee.colorado.edu/~siewerts/extra/code/example_code_archive/a490dmis_code/CUDA/cuda_work/samples/0_Simple/vectorAdd/vectorAdd.cu and https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#using-cuda-gpu-timers



#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>

#define n 1000 //job size = 1K, 10K, 100K, 1M and 10M

__global__ void add(int *a, int *b, int *c)  //add kernel
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

//function to generate random numbers 
void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%n;
	}
}

int main(void)
{

	int *a, *b, *c; // CPU copies 
	int *d_a, *d_b, *d_c; // GPU copies
	int size = n * sizeof(int);
	
	hipEvent_t start, stop; //time start and stop
	float time;

	hipEventCreate(&start); 
	hipEventCreate(&stop);

	//Allocate device memory
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	//Allocate CPU memory 
	a = (int *)malloc(size); random_ints(a, n);
	b = (int *)malloc(size); random_ints(b, n);
	c = (int *)malloc(size);

	hipEventRecord( start, 0 );
	
	//Copy CPU memory to GPU memory
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	
	//Call the add kernel
	add<<<1,n>>>(d_a, d_b, d_c); //1 thread block with n threads 

	//Copy from device to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	hipEventRecord( stop, 0 );
	hipEventSynchronize(stop);
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	printf("GPU Execution Time = %f\n",time);

	for (int i=0;i<n;i++) {
		printf("a[%d]=%d , b[%d]=%d, c[%d]=%d\n",i,a[i],i,b[i],i,c[i]);
	} //print the result
	
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}


