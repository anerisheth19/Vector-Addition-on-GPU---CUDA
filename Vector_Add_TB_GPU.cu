#include "hip/hip_runtime.h"
//ECGR 6090 Heterogeneous Computing Homework 0
// Problem 1 b - Vector Add on GPU
//Written by Aneri Sheth - 801085402

// Reference taken from Lecture Slides by Dr. Tabkhi 
// Other references taken from - http://ecee.colorado.edu/~siewerts/extra/code/example_code_archive/a490dmis_code/CUDA/cuda_work/samples/0_Simple/vectorAdd/vectorAdd.cu and https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#using-cuda-gpu-timers


#include<stdio.h>
#include<stdlib.h>
#include<time.h>

#define n 1000000 //fixed job size
#define m 128 //thread block size

__global__ void add(int *a, int *b, int *c, int k)  //add kernel
{
	int index = threadIdx.x+ blockIdx.x * blockDim.x
	if (index<k)
		c[index] = a[index] + b[index];
}

//function to get random numbers
void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%n;
	}
}

int main(void)
{

	int *a, *b, *c; //CPU copies
	int *d_a, *d_b, *d_c; //GPU copies
	int size = n * sizeof(int);

	hipEvent_t start, stop; //time start and stop
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Allocate device memory
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	//Allocate CPU memory
	a = (int *)malloc(size); random_ints(a, n);
	b = (int *)malloc(size); random_ints(b, n);
	c = (int *)malloc(size);

	hipEventRecord( start, 0 );
	
	//Copy CPU memory to GPU memory	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	
	//Call the add kernel
	add<<<(n+m-1)/m,m>>>(d_a, d_b, d_c,n); 
	
	printf("GPU Execution Time = %f\n,time);
	
	// Copy from device to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	hipEventRecord( stop, 0 );
	hipEventSynchronize(stop);
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Time = %f\n",time);


	for (int i=0;i<n;i++) {
		printf("a[%d]=%d , b[%d]=%d, c[%d]=%d\n",i,a[i],i,b[i],i,c[i]); 
	} //print the result
	
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;

}


